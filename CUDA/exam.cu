#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include <iostream>



using namespace std;



void checkCUBLAS(hipblasStatus_t status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
        cout << "[ERROR] CUBLAS " << status << endl;
}

void print(char* title, float* src, int h, int w)
{
    cout << title << endl;
    for (int y = 0; y < h; y++) 
    {
        for (int x = 0; x < w; x++) 
        {
            int index = y * w + x;
            printf("%5.0f", src[index]);
        }
        printf("\n");
    }
    printf("\n");
}


int main(int argc, char *argv[])
{

    const int N = 2;// Height
    const int M = 5;// Width
    float* a = new float[M * N];
    float* a_d;

    for (int i = 0; i < M*N; i++) 
        a[i] = i;

    print("src", a, N, M);
    checkCUBLAS(cublasInit());
    checkCUBLAS(cublasAlloc(M*N, sizeof(float), (void**)&a_d));
    checkCUBLAS(hipblasSetMatrix(M, N, sizeof(float), a, M, a_d, M));

    float alpha = 2;
    // a_d[j] = alpha * a_d[j]

    hipblasSscal(M*N, alpha, a_d, 1);
    checkCUBLAS(hipblasGetMatrix(M, N, sizeof(float), a_d, M, a, M));
    print("src * 2", a, N, M);

    hipDeviceSynchronize();
    checkCUBLAS(cublasFree(a_d));
    checkCUBLAS(cublasShutdown());

    delete []a;

    return 0;
}
