#include "hip/hip_runtime.h"
#include <stdio.h>
/*
#define N (1024*1024)
#define M (1000000)

__global__ void cudakernel(int *buf)
{
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   buf[i] = i;
   for(int j = 0; j < M; j++)
      buf[i] = buf[i] * buf[i] + 1;
}

int main()
{
   int data[N]; 
   int count = 0;
   int *d_data;
   hipMalloc(&d_data, N * sizeof(int));
   cudakernel<<<N/256, 256>>>(d_data);
   hipMemcpy(data, d_data, N * sizeof(int), hipMemcpyDeviceToHost);
   hipFree(d_data); 

   int sel;
   printf("Enter an index: ");
   scanf("%d", &sel);
   printf("data[%d] = %d\n", sel, data[sel]);
}
*/

#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include <iostream>



using namespace std;



void checkCUBLAS(hipblasStatus_t status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
        cout << "[ERROR] CUBLAS " << status << endl;
}

void print(char* title, float* src, int h, int w)
{
    cout << title << endl;
    for (int y = 0; y < h; y++) 
    {
        for (int x = 0; x < w; x++) 
        {
            int index = y * w + x;
            printf("%5.0f", src[index]);
        }
        printf("\n");
    }
    printf("\n");
}


int main(int argc, char *argv[])
{

    const int N = 2;// Height
    const int M = 5;// Width
    float* a = new float[M * N];
    float* a_d;

    for (int i = 0; i < M*N; i++) 
        a[i] = i;

    print("src", a, N, M);
    checkCUBLAS(cublasInit());
    checkCUBLAS(cublasAlloc(M*N, sizeof(float), (void**)&a_d));
    checkCUBLAS(hipblasSetMatrix(M, N, sizeof(float), a, M, a_d, M));

    float alpha = 2;
    // a_d[j] = alpha * a_d[j]

    hipblasSscal(M*N, alpha, a_d, 1);
    checkCUBLAS(hipblasGetMatrix(M, N, sizeof(float), a_d, M, a, M));
    print("src * 2", a, N, M);

    hipDeviceSynchronize();
    checkCUBLAS(cublasFree(a_d));
    checkCUBLAS(cublasShutdown());

    delete []a;

    return 0;
}