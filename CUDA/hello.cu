#include <stdio.h>
#include <iostream>
#include <hipblas.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include ""
using namespace std;

#define N 4
int main(){
    hipblasHandle_t handle;
    float arr1[] = {1,0,1,0,0,1,0,2,0,0,1,0,2,2,0,1};
    float arr2[] = {1,0,1,0,0,1,0,2,0,0,1,0,2,2,0,1};
    float arr3[] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    float* arr1_d;
    float* arr2_d;
    float* arr3_d;

    hipblasCreate(&handle);
    cublasAlloc(N*N, sizeof(float), (void**)&arr1_d);
    cublasAlloc(N*N, sizeof(float), (void**)&arr2_d);
    cublasAlloc(N*N, sizeof(float), (void**)&arr3_d);
    hipblasSetMatrix(N, N, sizeof(float), arr1, N, arr1_d, N);
    hipblasSetMatrix(N, N, sizeof(float), arr2, N, arr2_d, N);
    float alpha = 1;
    float beta = 0;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, arr1_d, N, arr2_d, N, &beta, arr3_d, N);

    hipblasGetMatrix(N, N, sizeof(float), arr3_d, N, arr3, N);

    hipDeviceSynchronize();
    cublasFree(arr1_d);
    cublasFree(arr2_d);
    cublasFree(arr3_d);

    cublasShutdown();
    for(int i=0; i<16; i++)
    	printf("%f",arr3_d[i]);

    return 0;
}
