#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 4
#define n 4
#define k 4
extern "C" void mat_mul_func(int size, float *a, float *b, float *c){

	hipError_t cudaStat;	//cudaMalloc status
	hipblasStatus_t stat;	//CUBLAS functions status
	hipblasHandle_t handle;	//CUBLAS context

	int i, j;
	

	//on the device
	float* d_a;
	float* d_b;
	float* d_c;

	cudaStat = hipMalloc((void**)&d_a, m*k*sizeof(a));
	cudaStat = hipMalloc((void**)&d_b, k*n*sizeof(b));
	cudaStat = hipMalloc((void**)&d_c, m*n*sizeof(c));
	
	stat = hipblasCreate(&handle);	//initialize CUBLAS context

	//copy matrixes from the host to the device
	stat = hipblasSetMatrix(m, k, sizeof(a), a, m, d_a, m);
	stat = hipblasSetMatrix(k, n, sizeof(*b), b, k, d_b, k);
	stat = hipblasSetMatrix(m, n, sizeof(*c), c, m, d_c, m);

	float al = 1.0f;
	float bet = 0.0f;

	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, d_a, m, d_b, k, &bet, d_c, m);

	stat = hipblasGetMatrix(m, n, sizeof(*c), d_c, m, c, m);
	printf("c after Sgemm : \n");
	for(i=0; i<m; i++){
		for(j=0; j<n; j++){
			printf("%7.0f", c[IDX2C(i, j, m)]);
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipblasDestroy(handle);


	
}
int main(){
	return 0;
}
