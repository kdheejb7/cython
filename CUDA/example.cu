#include "hip/hip_runtime.h"
// Kernel definition 
// Run on GPU

#include <stdio.h>
#define N 4
__global__ void add(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[1] = a[1] + b[1];
}

void mat_mul_func(int size, float *arr1, float *arr2, float *arr3) {
	
    float *d_a, *d_b, *d_c; // device copies of a, b, c
    // Allocate space for device copies of a, b, c
    int _size = sizeof(float)*size;
    hipMalloc((void **)&d_a, _size);
    hipMalloc((void **)&d_b, _size);
    hipMalloc((void **)&d_c, _size);
	
    // Copy a & b from the host to the device
    hipMemcpy(d_a, &a, _size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, _size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, _size, hipMemcpyHostToDevice);
	
    // Launch add() kernel on GPU
    add<<<N,1>>>(arr3, arr1, arr2);
	
    // Copy result back to the host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    for (int i = 0; i<size; i++)
	    printf("%d", c[i]);
    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}

/*
#include <stdio.h>
__global__ void add(int *a){
    *a = *a+2;
}
int main(void){
    int a = 3;
    int *d_a;

    hipMalloc((void **)&d_a, sizeof(int));
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    add<<<1,1>>>(d_a);
    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d",a);
    hipFree(d_a);
    return 0;
}
*/

//그래픽 카드에 메모리 사용하기
/*
#include <stdio.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main()
{
    int InputData[5] = {1, 2, 3, 4, 5};
    int OutputData[5] = {0};
 
    int* GraphicsCard_memory;
 
    //그래픽카드 메모리의 할당
    gpuErrchk( hipMalloc(&GraphicsCard_memory, 5*sizeof(int)) );

    //PC에서 그래픽 카드로 데이터 복사
    hipMemcpy(GraphicsCard_memory, InputData, 5*sizeof(int), hipMemcpyHostToDevice);
 
    //그래픽 카드에서 PC로 데이터 복사
    hipMemcpy(OutputData, GraphicsCard_memory, 5*sizeof(int), hipMemcpyDeviceToHost);
 
    //결과 출력
    for( int i = 0; i < 5; i++)
    {
        printf(" OutputData[%d] : %d\n", i, OutputData[i]);
    }
 
    //그래픽 카드 메모리의 해체
    hipFree(GraphicsCard_memory);
 
    return 0;
}
*/